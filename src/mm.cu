#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <mm_malloc.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define GRID_SIZE 150
using namespace std;
hipEvent_t start, stop;

int validate(float *c, int n) {
    //all resulting columns should have the same values
    for(unsigned i = 0; i < n*n ; i += n) {
        float tmp = c[i];
        for(unsigned j = 0; j < n; j++) {
            if(c[i + j] != tmp) return 0;
        }
    }
    return 1;
}
void startStopWatch () {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
}

void stopStopWatch () {
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time = 0;
	hipEventElapsedTime(&time, start, stop);
	cout << time << " ms." << endl;
}

__global__ void kernel_mm3(float *d_a, float *d_b, float *d_result, int n)
{
    float tile_a[BLOCK_SIZE];
    float tile_b[BLOCK_SIZE];
    int gs = GRID_SIZE * BLOCK_SIZE;
    int gg = GRID_SIZE * GRID_SIZE;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float tmp = 0;

    for (size_t i = 0; i < GRID_SIZE; i++)
    {
        tile_a[tx * BLOCK_SIZE + ty] = d_a[i * BLOCK_SIZE + tx * gs + ty];
        tile_b[tx * BLOCK_SIZE + ty] = d_b[i * gg + tx * gs + ty];
        __syncthreads();
        for (int i=0; i<BLOCK_SIZE; i++)
        {
            tile_a[i] += __shfl_sync(-1, tile_a[i], i);
            tile_b[i] += __shfl_sync(-1, tile_b[i], i);
        }

        for (size_t i = 0; i < BLOCK_SIZE; i++)
        {
            tmp += tile_a[tx * i + i] * tile_b[ty * i + i];
        }
    }

    d_result[tx * n + ty] = tmp;
}



__global__ void kernel_mm2(float *d_a, float *d_b, float *d_result, int n)
{
    __shared__ float tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if (idx >= n * n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if (idx >= n * n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

__global__ void kernel_mm(float *d_a, float *d_b, float *d_result, int n)
{
    __shared__ float tile_a[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float tile_b[BLOCK_SIZE * BLOCK_SIZE];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float tmp = 0;





    int startA = by * n * BLOCK_SIZE; // 0 
    int stepA  = BLOCK_SIZE;
    int endA   = by * n * BLOCK_SIZE + n; // 8 
    int startB = bx * BLOCK_SIZE;
    int stepB  = BLOCK_SIZE * n;
    int endB   = BLOCK_SIZE * stepB; 
    
    for (size_t a = startA, size_t b = startB; a < endA; a+=stepA, b+=stepB)
    {
        tile_a[ty * BLOCK_SIZE + tx] = d_a[a+tx+ty*n];
        tile_b[ty * BLOCK_SIZE + tx] = d_b[b+tx+ty*n];
        __syncthreads();
        for (size_t i = 0; i < BLOCK_SIZE; i++)
        {
            tmp += tile_a[ty * BLOCK_SIZE + i] * tile_b[i * BLOCK_SIZE + tx];
        }
	__syncthreads();
    }

    d_result[tx * n + ty] = tmp;
}

__global__
void multMat(float *a,float *b, float *c, int N){
    int linha=blockIdx.y*blockDim.y+threadIdx.y;
    int coluna=blockIdx.x*blockDim.x+threadIdx.x;
    float sum=0;
    if(coluna<N&&linha<N){
        for(int i=0;i<N;i++)sum+=a[linha*N+i]*b[i*N+coluna];
        c[linha*N+coluna]=sum;
    }
}


void checker(float *c, int N){
    if (hipSuccess==hipGetLastError() && validate(c,N)){
        cout << "NO ERROR" << endl;
    }
    else{
        cout << "There was an error" << endl;
    }
}
void stencil(float *a, float *b, float *c, int N){
    float *devA,*devB, *devC;
    int NQ = N*N;
    hipMalloc((void**) &devA, NQ * sizeof(float));
    hipMalloc((void**) &devB, NQ * sizeof(float));
    hipMalloc((void**) &devC, NQ * sizeof(float));

    startStopWatch();
	hipMemcpy(devA,a,NQ*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(devB,b,NQ*sizeof(float),hipMemcpyHostToDevice);
    stopStopWatch();
    dim3 dimGrid(150,150);
    dim3 dimBlock(16,16);
    startStopWatch();
    kernel_mm3<<<dimGrid,dimBlock>>>(devA,devB,devC,N);
    stopStopWatch();
    startStopWatch();
    hipMemcpy(c,devC,NQ*sizeof(float),hipMemcpyDeviceToHost);
    stopStopWatch();
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void newMatrices(float **a, float **b, float **c, int N){
    int i;
    int NQ = N*N;
    *a = (float *)_mm_malloc(NQ * sizeof(float), 32);
    *b = (float *)_mm_malloc(NQ * sizeof(float), 32);
    *c = (float *)_mm_malloc(NQ * sizeof(float), 32);
    for (i = 0; i < NQ; i++){
        (*b)[i] = 1;
        (*a)[i] = ((float)rand()) / ((float)RAND_MAX);
    }
}

int main (int argc, char** argv) {
  	int N = atoi(argv[1]);
    srand(0);
	float *a,*b,*c;
    newMatrices(&a,&b,&c,N);
    stencil(a,b,c,N);
    checker(c,N);
	return 0;
}