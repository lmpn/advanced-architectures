
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <mm_malloc.h>
#include <stdio.h>  /* printf, scanf, puts, NULL */
#include <stdlib.h> /* srand, rand */
#include <sys/time.h>
#include <time.h> /* time */

using namespace std;
int f = 0;
#define BLOCK_SIZE 32
#define GRID_SIZE 75

hipEvent_t start, stop;

void startKernelTime(void)
{
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

void stopKernelTime(void)
{
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        cerr << "Cuda error: " << msg << ", " << hipGetErrorString(err) << endl;
        exit(-1);
    }
}

__global__ void matrixMultKernel(float *a, float *b, float *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < n && row < n)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}
__global__ void kernel_mm(float *d_a, float *d_b, float *d_result, int n)
{
     __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}


__global__ void kernel_mm2(float *d_a, float *d_b, float *d_result, int n)
{
    float tile_a[BLOCK_SIZE];
    float tile_b[BLOCK_SIZE];
    int gs = GRID_SIZE * BLOCK_SIZE;
    int gg = GRID_SIZE * GRID_SIZE;
    int tx = threadIdx.y;
    int ty = threadIdx.x;
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float tmp = 0;
    
    for (size_t i = 0; i < GRID_SIZE; i++)
    {
        tile_a[tx * BLOCK_SIZE + ty] = d_a[i * BLOCK_SIZE + tx * gs + ty];
        tile_b[tx * BLOCK_SIZE + ty] = d_b[i * gg + tx * gs + ty];
        __syncthreads();
        for (int i=0; i<BLOCK_SIZE; i++)
        {
            tile_a[i] += __shfl(-1, tile_a[i], i);
            tile_b[i] += __shfl(-1, tile_b[i], i);
        }
        
        for (size_t i = 0; i < BLOCK_SIZE; i++)
        {
            tmp += tile_a[tx * i + i] * tile_b[ty * i + i];
        }
    }

    d_result[row * n + col] = tmp;
}

__global__ void bmatrixMultKernel(float *d_a, float *d_b, float *d_result, int n)
{
    __shared__ float tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if (idx >= n * n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if (idx >= n * n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

float *stencilGPU(float *a, float *b, int size)
{
    float *dev_a, *dev_b, *dev_c;
    float *c = new float[size * size];

    hipMalloc((void **)&dev_a, size * size * sizeof(float));
    hipMalloc((void **)&dev_b, size * size * sizeof(float));
    hipMalloc((void **)&dev_c, size * size * sizeof(float));

    startKernelTime();
    hipMemcpy(dev_a, a, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * size * sizeof(float), hipMemcpyHostToDevice);
    stopKernelTime();

    startKernelTime();
    if (f == 0)
    {
        dim3 dimGrid(size, size);
        dim3 dimBlock(1, 1);
        matrixMultKernel<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, size);
    }
    else
    {
        dim3 dimGrid(75, 75);
        dim3 dimBlock(32, 32);
        // bmatrixMultKernel <<< dimGrid, dimBlock >>>(dev_a, dev_b, dev_c, size);
        kernel_mm<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, size);
    }
    stopKernelTime();

    startKernelTime();
    hipMemcpy(c, dev_c, size * size * sizeof(float), hipMemcpyDeviceToHost);
    stopKernelTime();

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for(size_t i = 0; i < 2400; i++)
    {
       for(size_t j = 0; j < 2400; j++)
       {
           printf("%lf ", c[i*2400 + j]);
       }
        printf("\n");
    }
    

    return c;
}

int init_matrices(float **a, float **b, int N)
{
    int i;
    const int total_elements = N * N;
    *a = (float *)_mm_malloc(N * N * sizeof(float), 32);
    *b = (float *)_mm_malloc(N * N * sizeof(float), 32);
    for (i = 0; i < total_elements; i++)
    {
        (*b)[i] = 1;
        (*a)[i] = ((float)rand()) / ((float)RAND_MAX);
    }
    return 1;
}

int main(int argc, char **argv)
{
    int size = atoi(argv[1]);
    f = atoi(argv[2]);
    printf("%d ", f);
    float *a, *b;
    init_matrices(&a, &b, size);
    for (int i = 0; i < 1; i++)
    {
        stencilGPU(a, b, size);
        printf("i = %d\n", i);
    }
    return 0;
}
